
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiplication(int* M, int* N, int* P, int Width)
{
    int tid, tx, ty;
	tx = blockDim.x*blockIdx.x + threadIdx.x;
	ty = blockDim.y*blockIdx.y + threadIdx.y;
	tid = Width*ty + tx;
       
  	int Value = 0;
	int MVal = 0;
	int NVal = 0;

	for (int i = 0; i < Width; i++)
	{
		MVal = M[ty * Width + i];
		NVal = N[i * Width + tx];
		Value += MVal * NVal;
	}

	P[tid] = Value;

}
